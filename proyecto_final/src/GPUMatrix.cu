#include "hip/hip_runtime.h"
#include <GPUMatrix/GPUMatrix.h>
#include <stdio.h>

namespace {
    __global__
    void H() {
        printf("Hello World from GPU! %d %d\n", blockIdx.x, threadIdx.x);
    }
}

namespace GPUMatrix {
    
void HelloThreadIdx() { H<<<2, 4>>>(); }

}  // namespace GPUMatrix